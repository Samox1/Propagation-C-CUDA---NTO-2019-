#include "hip/hip_runtime.h"
// Kappa - this is project file for NTO project - Light Propagation with GPU

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

using namespace std;
// --- Main Part ---

/*
 * complie: nvcc -o prop.x prop.cu -O3 -gencode=arch=compute_35,code=sm_35 -gencode=arch=compute_37,code=sm_37 -gencode=arch=compute_60,code=sm_60 -I/usr/local/cuda/inc -L/usr/local/cuda/lib -lcufft
 */
int main(int *argc, char *argv[])
{
	ifstream inputFile;
	int COL = atoi(argv[2]);
	int ROW = atoi(argv[3]);
	float u_in[ROW][COL];
	cout << "DUPA WELCOME" << " | " << argv[0] << " | " << argv[1] << " | " << argv[2] << " | " << argv[3] << endl;
	cout << "ROW: " << ROW << " | " << "COL: " << COL <<endl;
	inputFile.open(argv[1]);
	if (inputFile)
	{
		int i,j = 0;
		for (i = 0; i < ROW; i++)
		{
			for (j = 0; j < COL; j++)
			{	
				inputFile >> u_in[i][j];
				//cout << u_in[i][j];
			}
		//cout << endl;
		}
		cout << endl;
	} else {
		cout << "Error opening the file.\n";
	}
	inputFile.close();
				// --- FFT tablicy wejsciowej --- //
	int NX = 2*COL;
	int NY = 2*ROW;
	hipfftHandle plan;
	
	hipfftComplex* data;
	data = (hipfftComplex *) malloc ( sizeof(hipfftComplex)* NX * NY);

	hipfftComplex* dData;
	hipMalloc((void **) &dData, sizeof(hipfftComplex)* NX * NY);

	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;
	}

	for(int ii=0; ii < NY ; ii++)
	{
		for(int jj=0; jj < NX ; jj++)
		{
			data[ii*NX+jj].x = 0;
			data[ii*NX+jj].y = 0;
		}
	}

	for(int ii=0; ii < (int)NY/2 ; ii++)
	{
		for(int jj=0; jj < (int)NX/2 ; jj++)
		{
			data[(ii*NX+jj)+(NX*NY/4+NX/4)].x = (float)u_in[ii][jj];
		}
	}
/*
	printf( "Org vals: \n");
	for(int ii=0; ii<NX*NY ; ii++)
	{
		if (ii%NX == 0){
			printf("\n");
		}
		printf ( "%.0f ", data[ii].x );	
	}
*/
	size_t pitch;
 	hipMallocPitch(&dData, &pitch, sizeof(hipfftComplex)*NX, NY);
	hipMemcpy2D(dData,pitch,data,sizeof(hipfftComplex)*NX,sizeof(hipfftComplex)*NX,NX,hipMemcpyHostToDevice);
 	
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}

// Create a 2D FFT plan. 
	if (hipfftPlan2d(&plan, NX, NY, HIPFFT_C2C) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		return -1;	
	}

	if (hipfftExecC2C(plan, dData, dData, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
		return -1;		
	}

	if (hipDeviceSynchronize() != hipSuccess){
  		fprintf(stderr, "Cuda error: Failed to synchronize\n");
   		return -1;
	}	

	hipMemcpy(data, dData, sizeof(hipfftComplex)*NX*NY, hipMemcpyDeviceToHost);
	
	printf( "\nCUFFT vals: \n");
	
// TEST - wypisania
	/*
	for(int ii=0; ii<NX*NY; ii++)
	{	
		data[ii].x = ii;
	}
	*/
// KONIEC TESTU
/*	int kappa = 0;
	for(int ii=NX*NY/4+NX/4; ii<(NX*NY)-(NX*NY/4+NX/4); ii++)
	{	
		data[ii].x = sqrt(pow(data[ii].x, 2) + pow(data[ii].y, 2));
		kappa++;
		if (kappa == NX/2){ ii += NX/2; kappa = 0;}
	}
	
	float mini_data = data[NX*NY/4+NX/4].x;
	

	kappa = 0;
	for(int ii=NX*NY/4+NX/4; ii<(NX*NY)-(NX*NY/4+NX/4); ii++)
	{
		if (data[ii].x < mini_data){ mini_data = data[ii].x; }
		kappa++;
		if (kappa == NX/2){ ii += NX/2; kappa = 0;}
	}
	
	float max_data = data[NX*NY/4+NX/4].x;
	kappa = 0;
	for(int ii=NX*NY/4+NX/4; ii<(NX*NY)-(NX*NY/4+NX/4); ii++)
	{
		data[ii].x = data[ii].x + abs(mini_data);
		if (data[ii].x > max_data) { max_data = data[ii].x; }
		kappa++;
		if (kappa == NX/2){ ii += NX/2; kappa = 0;}
	}

	kappa = 0;
	for(int ii=NX*NY/4+NX/4; ii<(NX*NY)-(NX*NY/4+NX/4); ii++)
	{	
		data[ii].x = data[ii].x / max_data * 255;
		//if (ii%NX/2 == 0){printf("\n");}
		printf ( "%.0f ", data[ii].x);
		
		kappa++;
		if (kappa == NX/2){ ii += NX/2; kappa = 0; printf("\n");}
	}
*/

// Czytanie calosci

	for(int ii=0; ii<(NX*NY); ii++)
	{	
		data[ii].x = sqrt(pow(data[ii].x, 2) + pow(data[ii].y, 2));
	}
	
	float mini_data = data[0].x;
	
	for(int ii=0; ii<(NX*NY); ii++)
	{		
		if (data[ii].x < mini_data){ mini_data = data[ii].x; }
	}
	
	float max_data = data[0].x;

	for(int ii=0; ii<(NX*NY); ii++)
	{		
		data[ii].x = data[ii].x + abs(mini_data);
		if (data[ii].x > max_data) { max_data = data[ii].x; }
	}

	for(int ii=0; ii<(NX*NY); ii++)
	{	
		if (ii%NX == 0){printf("\n");}
		data[ii].x = data[ii].x / max_data * 255;
		printf ( "%.0f ", data[ii].x);
	}


	hipfftDestroy(plan);
	hipFree(data);
	hipFree(dData);
	return 0;
} 
