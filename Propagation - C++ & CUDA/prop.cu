#include "hip/hip_runtime.h"
// Kappa - this is project file for NTO project - Light Propagation with GPU
// Autorzy: Szymon Baczyński && Łukasz Szeląg
// Projekt na przedmiot NTO 2018/2019

#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>

using namespace std;


__global__ void multiplyElementwise(hipfftDoubleComplex* f0, hipfftDoubleComplex* f1, int size)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < size)
    {
        double a, b, c, d;
        a = f0[i].x; 
        b = f0[i].y;
        c = f1[i].x; 
        d = f1[i].y;
        f0[i].x = a*c - b*d;
        f0[i].y = a*d + b*c;
    }
}


void h_z(double lam, double z, double k, double sampling, int NX, int NY, hipfftDoubleComplex* h_z_cutab)
{
	std::complex<double>* h_z_tab;
	h_z_tab = (std::complex<double> *) malloc ( sizeof(std::complex<double>)* NX * NY);

	double fi = k * z;
	double teta = k / (2.0 * z);
	double lam_z = lam * z;
	double quad = 0.0;
	double teta1 = 0.0;	

	for(int iy=0; iy < NY; iy++)
	{
		//printf("\n");
		for(int ix=0; ix < NX ; ix++)
		{
			quad = pow(((double)ix-((double)NX/2.0))*sampling, 2) + pow(((double)iy-((double)NY/2.0))*sampling, 2);
			teta1 = teta * quad;
			h_z_tab[iy*NX+ix] = exp(1i*fi)*exp(1i*teta1)/(1i*lam_z);
			h_z_cutab[iy*NX+ix].x = h_z_tab[iy*NX+ix].real();
			h_z_cutab[iy*NX+ix].y = h_z_tab[iy*NX+ix].imag();
			//printf("%.2f\t", h_z_cutab[iy*NX+ix].x);
		}
	}	
	free(h_z_tab);
}


void Q_roll(hipfftDoubleComplex* u_in_fft, hipfftDoubleComplex* data, int NX, int NY)
{
	for(int iy=0; iy<(NY/4); iy++)	//Petla na przepisanie tablicy koncowej
	{
		for(int jx=0; jx<(NX/4); jx++)
		{
			u_in_fft[(NX/2*NY/4+NY/4)+(jx+iy*NX/2)] = data[iy*(NX)+jx];		// Q1 -> Q4
			u_in_fft[(jx+NX/4)+(iy*NX/2)] = data[(iy*(NX)+jx)+(NX*NY*3/4)];		// Q3 -> Q2
			u_in_fft[(jx)+(iy*NX/2)] = data[((iy*NX)+jx)+(NX*3/4+NX*NY*3/4)];	// Q4 -> Q1
			u_in_fft[(jx)+(iy*NX/2)+NX*NY/2/4] = data[((iy*NX)+jx)+(NX*3/4)];	// Q2 -> Q3
		}
	}
}

void amplitude_print(hipfftDoubleComplex* u_in_fft, int NX, int NY)
{
	// --- Przeliczanie Amplitudy --- //

	for(int ii=0; ii<(NX*NY/4); ii++)
	{	
		u_in_fft[ii].x = sqrt(pow(u_in_fft[ii].x, 2) + pow(u_in_fft[ii].y, 2));
	}
	
	double mini_data = u_in_fft[0].x;
	
	for(int ii=0; ii<(NX*NY/4); ii++)
	{		
		if (u_in_fft[ii].x < mini_data){ mini_data = u_in_fft[ii].x; }
	}
	
	double max_data = u_in_fft[0].x;
	mini_data = -mini_data;
	
	for(int ii=0; ii<(NX*NY/4); ii++)
	{		
		u_in_fft[ii].x = u_in_fft[ii].x + mini_data;
		if (u_in_fft[ii].x > max_data) { max_data = u_in_fft[ii].x; }
	}

	for(int ii=0; ii<(NX*NY/4); ii++)
	{	
		if (ii%(NX/2) == 0){printf("\n");}
		u_in_fft[ii].x = u_in_fft[ii].x / max_data * 255.0;
		printf ("%.0f\t", u_in_fft[ii].x);
	}
}

int FFT_Z2Z(hipfftDoubleComplex* dData, int NX, int NY)
{
	// Create a 2D FFT plan. 
	int err = 0;
	hipfftHandle plan1;
	if (hipfftPlan2d(&plan1, NX, NY, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		err = -1;	
	}

	if (hipfftExecZ2Z(plan1, dData, dData, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
		err = -1;		
	}

	if (hipDeviceSynchronize() != hipSuccess){
  		fprintf(stderr, "Cuda error: Failed to synchronize\n");
   		err = -1;
	}	
	
	hipfftDestroy(plan1);
	return err;
}

int IFFT_Z2Z(hipfftDoubleComplex* dData, int NX, int NY)
{
	// Create a 2D FFT plan.
	int err = 0; 
	hipfftHandle plan1;
	if (hipfftPlan2d(&plan1, NX, NY, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		err = -1;	
	}

	if (hipfftExecZ2Z(plan1, dData, dData, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
		err = -1;		
	}

	if (hipDeviceSynchronize() != hipSuccess){
  		fprintf(stderr, "Cuda error: Failed to synchronize\n");
   		err = -1;
	}

	hipfftDestroy(plan1);	
	return err;
}

/*
 * complie: nvcc -o prop.x prop.cu -O3 -gencode=arch=compute_35,code=sm_35 -gencode=arch=compute_37,code=sm_37 -gencode=arch=compute_60,code=sm_60 -I/usr/local/cuda/inc -L/usr/local/cuda/lib -lcufft
 * start program: ./prop.x Tablica-1024x1024.txt 1024 1024 > 1024x1024.txt
 */

// --- Main Part --- //

int main(int *argc, char *argv[])
{
	ifstream inputFile;
	int COL = atoi(argv[2]);
	int ROW = atoi(argv[3]);
	double u_in[ROW][COL];
	cout << "DUPA WELCOME" << " | " << argv[0] << " | " << argv[1] << " | " << argv[2] << " | " << argv[3] << endl;
	cout << "ROW: " << ROW << " | " << "COL: " << COL <<endl;
	inputFile.open(argv[1]);
	if (inputFile)
	{
		int i,j = 0;
		for (i = 0; i < ROW; i++)
		{
			for (j = 0; j < COL; j++)
			{	
				inputFile >> u_in[i][j];
			}
		}
		cout << endl;
	} else {
		cout << "Error opening the file.\n";
	}
	inputFile.close();


// --- Liczenie propagacji i FFT --- //

	int NX = 2*COL;
	int NY = 2*ROW;

// --- Przeliczenie h_z --- //

	double sampling = 10.0 * pow(10.0, (-6)); 	// Sampling = 10 micro
	double lam = 633.0 * (pow(10.0,(-9))); 		// Lambda = 633 nm
	double k = 2.0 * M_PI / lam;			// Wektor falowy k
	double z = 1000.0*(pow(10.0,(-3)));		// Odleglosc propagacji = 1 metr

	printf("k = %.1f | lam = %.1f | z = %.1f", k, lam, z);

	hipfftDoubleComplex* h_z_tab;
	h_z_tab = (hipfftDoubleComplex *) malloc ( sizeof(hipfftDoubleComplex)* NX * NY);
	h_z(lam, z, k, sampling, NX, NY, h_z_tab);	// Liczenie h_z

	printf("\n");


// --- FFT tablicy wejsciowej --- //
	
	hipfftDoubleComplex* data;
	data = (hipfftDoubleComplex *) malloc ( sizeof(hipfftDoubleComplex)* NX * NY);

	hipfftDoubleComplex* dData;
	hipMalloc((void **) &dData, sizeof(hipfftDoubleComplex)* NX * NY);

	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;
	}

	for(int ii=0; ii < NY ; ii++)
	{
		for(int jj=0; jj < NX ; jj++)
		{
			data[ii*NX+jj].x = 0;
			data[ii*NX+jj].y = 0;
		}
	}

	for(int ii=0; ii < (int)NY/2 ; ii++)
	{
		for(int jj=0; jj < (int)NX/2 ; jj++)
		{
			data[(ii*NX+jj)+(NX*NY/4+NX/4)].x = (double)u_in[ii][jj];
		}
	}

// Liczenie U_in = FFT{u_in}
	
	size_t pitch1;
 	hipMallocPitch(&dData, &pitch1, sizeof(hipfftDoubleComplex)*NX, NY);
	hipMemcpy2D(dData,pitch1,data,sizeof(hipfftDoubleComplex)*NX,sizeof(hipfftDoubleComplex)*NX,NX,hipMemcpyHostToDevice);
 	
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}

	if (FFT_Z2Z(dData, NX, NY) == -1) { return -1; }

// --- Liczenie H_Z = FFT{h_z_tab} --- //
	
	hipfftDoubleComplex* H_Z;
	hipMalloc((void **) &H_Z, sizeof(hipfftDoubleComplex)* NX * NY);

	size_t pitch2;
 	hipMallocPitch(&H_Z, &pitch2, sizeof(hipfftDoubleComplex)*NX, NY);
	hipMemcpy2D(H_Z,pitch2,h_z_tab,sizeof(hipfftDoubleComplex)*NX,sizeof(hipfftDoubleComplex)*NX,NX,hipMemcpyHostToDevice);
 	
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}

	if (FFT_Z2Z(H_Z, NX, NY) == -1) { return -1; }

	// Do the actual multiplication

	multiplyElementwise<<<NX*NY, 1>>>(dData, H_Z, NX*NY);
	

// --- Liczenie u_out = iFFT{dData = U_OUT} --- //

	if (IFFT_Z2Z(dData, NX, NY) == -1) { return -1; }


	hipMemcpy(data, dData, sizeof(hipfftDoubleComplex)*NX*NY, hipMemcpyDeviceToHost);

	printf( "\nCUFFT vals: \n");
	
//TEST - wypisania
//Test do kasacji	
/*	int NX = 12;		//Pomoc
	int NY = 12;		//Test na mniejszej tablicy

	hipfftComplex* data;
	data = (hipfftDoubleComplex *) malloc ( sizeof(hipfftDoubleComplex)* NX * NY);

	for(int ii=0; ii<NX*NY; ii++)
	{	
		data[ii].x = ii;
		data[ii].y = ii;
	}
*/	
//KONIEC TESTU

// Czytanie calosci


// --- ROLL cwiartek, zeby wszystko sie zgadzalo na koniec --- //

	hipfftDoubleComplex* u_out;
	u_out = (hipfftDoubleComplex *) malloc (sizeof(hipfftDoubleComplex)* NX/2 * NY/2);

	Q_roll(u_out, data, NX, NY);

// --- Przeliczanie Amplitudy --- //

	amplitude_print(u_out, NX, NY);
		
	hipFree(u_out);
	hipFree(data);
	hipFree(dData);
	hipFree(h_z_tab);
	hipFree(H_Z);

	return 0;
} 
