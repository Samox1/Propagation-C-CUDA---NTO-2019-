#include "hip/hip_runtime.h"
// Kappa - this is project file for NTO project - Light Propagation with GPU
// Autorzy: Szymon Baczyński && Łukasz Szeląg
// Projekt na przedmiot NTO 2018/2019

#ifdef _WIN32
#define WIN32_LEAN_AND_MEAN
#include <windows.h>
#else
#include <unistd.h>
#endif
#include <iostream>
#include <fstream>
#include <stdio.h>
#include <math.h>
#include <complex>
#include <cmath>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipfft/hipfft.h>
#include <omp.h>
#include <mpi.h>

using namespace std;


__global__ void multiplyElementwise(hipfftDoubleComplex* f0, hipfftDoubleComplex* f1, int size)
{
    const int i = blockIdx.x*blockDim.x + threadIdx.x;
    if (i < size)
    {
        double a, b, c, d;
        a = f0[i].x; 
        b = f0[i].y;
        c = f1[i].x; 
        d = f1[i].y;
        f0[i].x = a*c - b*d;
        f0[i].y = a*d + b*c;
    }
}

void u_in_in_big(double* u_in, hipfftDoubleComplex* data, int NX, int NY, int multi)
{
	for(int ii=0; ii < NY ; ii++)
	{
		for(int jj=0; jj < NX ; jj++)
		{
			data[ii*NX+jj].x = 0;
			data[ii*NX+jj].y = 0;
		}
	}

	for(int ii=0; ii < (int)NY/multi ; ii++)
	{
		for(int jj=0; jj < (int)NX/multi ; jj++)
		{
			data[(ii*NX+jj)+(NX*NY*(multi-1)/(multi*2)+NX*(multi-1)/(multi*2))].x = u_in[ii*(NX/multi)+jj];
		}
	}
}


void h_z(double lam, double z, double k, double sampling, int NX, int NY, hipfftDoubleComplex* h_z_cutab)
{
	std::complex<double>* h_z_tab;
	h_z_tab = (std::complex<double> *) malloc ( sizeof(std::complex<double>)* NX * NY);

	double fi = k * z;
	double teta = k / (2.0 * z);
	double lam_z = lam * z;
	double quad = 0.0;
	double teta1 = 0.0;	

	for(int iy=0; iy < NY; iy++)
	{
		//printf("\n");
		for(int ix=0; ix < NX ; ix++)
		{
			quad = pow(((double)ix-((double)NX/2.0))*sampling, 2) + pow(((double)iy-((double)NY/2.0))*sampling, 2);
			teta1 = teta * quad;
			h_z_tab[iy*NX+ix] = exp(1i*fi)*exp(1i*teta1)/(1i*lam_z);
			h_z_cutab[iy*NX+ix].x = h_z_tab[iy*NX+ix].real();
			h_z_cutab[iy*NX+ix].y = h_z_tab[iy*NX+ix].imag();
			//printf("%.2f\t", h_z_cutab[iy*NX+ix].x);
		}
	}	
	free(h_z_tab);
}


void Q_roll(hipfftDoubleComplex* u_in_fft, hipfftDoubleComplex* data, int NX, int NY)
{
	for(int iy=0; iy<(NY/4); iy++)	//Petla na przepisanie tablicy koncowej
	{
		for(int jx=0; jx<(NX/4); jx++)
		{
			u_in_fft[(NX/2*NY/4+NY/4)+(jx+iy*NX/2)] = data[iy*(NX)+jx];		// Q1 -> Q4
			u_in_fft[(jx+NX/4)+(iy*NX/2)] = data[(iy*(NX)+jx)+(NX*NY*3/4)];		// Q3 -> Q2
			u_in_fft[(jx)+(iy*NX/2)] = data[((iy*NX)+jx)+(NX*3/4+NX*NY*3/4)];	// Q4 -> Q1
			u_in_fft[(jx)+(iy*NX/2)+NX*NY/2/4] = data[((iy*NX)+jx)+(NX*3/4)];	// Q2 -> Q3
		}
	}
}

void amplitude_print(hipfftDoubleComplex* u_in_fft, int NX, int NY, FILE* fp)
{
	// --- Przeliczanie Amplitudy --- //

	for(int ii=0; ii<(NX*NY/4); ii++)
	{	
		u_in_fft[ii].x = sqrt(pow(u_in_fft[ii].x, 2) + pow(u_in_fft[ii].y, 2));
	}
	
	double mini_data = u_in_fft[0].x;
	
	for(int ii=0; ii<(NX*NY/4); ii++)
	{		
		if (u_in_fft[ii].x < mini_data){ mini_data = u_in_fft[ii].x; }
	}
	
	double max_data = u_in_fft[0].x;
	mini_data = -mini_data;
	
	for(int ii=0; ii<(NX*NY/4); ii++)
	{		
		u_in_fft[ii].x = u_in_fft[ii].x + mini_data;
		if (u_in_fft[ii].x > max_data) { max_data = u_in_fft[ii].x; }
	}

	for(int ii=0; ii<(NX*NY/4); ii++)
	{	
		if (ii%(NX/2) == 0){fprintf (fp,"\n");}
		u_in_fft[ii].x = u_in_fft[ii].x / max_data * 255.0;
		fprintf (fp,"%.0f\t", u_in_fft[ii].x);
	}
}

int FFT_Z2Z(hipfftDoubleComplex* dData, int NX, int NY)
{
	// Create a 2D FFT plan. 
	int err = 0;
	hipfftHandle plan1;
	if (hipfftPlan2d(&plan1, NX, NY, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		err = -1;	
	}

	if (hipfftExecZ2Z(plan1, dData, dData, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
		err = -1;		
	}

	if (hipDeviceSynchronize() != hipSuccess){
  		fprintf(stderr, "Cuda error: Failed to synchronize\n");
   		err = -1;
	}	
	
	hipfftDestroy(plan1);
	return err;
}

int IFFT_Z2Z(hipfftDoubleComplex* dData, int NX, int NY)
{
	// Create a 2D FFT plan.
	int err = 0; 
	hipfftHandle plan1;
	if (hipfftPlan2d(&plan1, NX, NY, HIPFFT_Z2Z) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to create plan\n");
		err = -1;	
	}

	if (hipfftExecZ2Z(plan1, dData, dData, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
		fprintf(stderr, "CUFFT Error: Unable to execute plan\n");
		err = -1;		
	}

	if (hipDeviceSynchronize() != hipSuccess){
  		fprintf(stderr, "Cuda error: Failed to synchronize\n");
   		err = -1;
	}

	hipfftDestroy(plan1);	
	return err;
}

/*
 * complie: nvcc -o prop.x prop.cu -O3 -gencode=arch=compute_35,code=sm_35 -gencode=arch=compute_37,code=sm_37 -gencode=arch=compute_60,code=sm_60 -I/usr/local/cuda/inc -L/usr/local/cuda/lib -lcufft -I/opt/openmpi-gcc721-Cuda90/3.1.1/include -Xcompiler "-pthread -fPIC" -Xlinker "-Wl,-rpath -Wl,/opt/openmpi-gcc721-Cuda90/3.1.1/lib -Wl,--enable-new-dtags" -L/opt/openmpi-gcc721-Cuda90/3.1.1/lib -lmpi
 * compile: nvcc -o prop.x prop.cu -O3 -gencode=arch=compute_35,code=sm_35 -gencode=arch=compute_37,code=sm_37 -gencode=arch=compute_60,code=sm_60 -I/usr/local/cuda/inc -L/usr/local/cuda/lib -lcufft -I/opt/openmpi-gcc721-Cuda90/3.1.1/include -Xcompiler "-pthread -fPIC" -L/opt/openmpi-gcc721-Cuda90/3.1.1/lib -lmpi
 * start program: ./prop.x Tablica-1024x1024.txt 1024 1024 > 1024x1024.txt
 */

// --- Main Part --- //

int main(int argc, char *argv[])
{

// MPI --- Kappa

	int ip, np;
	MPI_Init(&argc, &argv);
    	MPI_Comm_rank(MPI_COMM_WORLD, &ip);
    	MPI_Comm_size(MPI_COMM_WORLD, &np);

	double starttime, endtime;
	starttime = MPI_Wtime();

	int COL = atoi(argv[2]);
	int ROW = atoi(argv[3]);
	double u_in[ROW*COL];

if (ip == 0)
{
	ifstream inputFile;
	//double u_in[ROW*COL];
	//cout << "DUPA WELCOME" << " | " << argv[0] << " | " << argv[1] << " | " << argv[2] << " | " << argv[3] << endl;
	//cout << "ROW: " << ROW << " | " << "COL: " << COL <<endl;
	inputFile.open(argv[1]);
	if (inputFile)
	{
		int i,j = 0;
		for (i = 0; i < ROW; i++)
		{
			for (j = 0; j < COL; j++)
			{	
				inputFile >> u_in[i*ROW+j];
			}
		}
	} else {
		cout << "Error opening the file.\n";
	}
	inputFile.close();
}

// --- Liczenie propagacji i FFT --- //
	int multi = atoi(argv[4]);
	int NX = COL*multi;
	int NY = ROW*multi;

// --- Przeliczenie h_z --- //

	double sampling = 10.0 * pow(10.0, (-6)); 	// Sampling = 10 micro
	double lam = 633.0 * (pow(10.0,(-9))); 		// Lambda = 633 nm
	double k = 2.0 * M_PI / lam;			// Wektor falowy k
	double z_in = 500.0*(pow(10.0,(-3)));		// Odleglosc propagacji = 1 metr
	double z_out = 1000.0*(pow(10.0,(-3)));
	double z_delta = 50.0*(pow(10.0,(-3)));
	double z = z_in+(ip*z_delta);
	
	//printf("k = %.1f | lam = %.1f | z = %.4f mm | ", k, lam*(pow(10.0,(9))), z);
	//printf(" ");
	//MPI_Barrier(MPI_COMM_WORLD);
	//printf("\n");

	//starttime = MPI_Wtime();
// --- FFT tablicy wejsciowej --- //
	hipfftDoubleComplex* data;
	data = (hipfftDoubleComplex *) malloc ( sizeof(hipfftDoubleComplex)* NX * NY);

	hipfftDoubleComplex* dData;
	hipMalloc((void **) &dData, sizeof(hipfftDoubleComplex)* NX * NY);

	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;
	}
	
	//starttime = MPI_Wtime();
	size_t pitch1;

if (ip == 0)
{
	u_in_in_big(u_in, data, NX, NY, multi);

	// Liczenie U_in = FFT{u_in}
 	hipMallocPitch(&dData, &pitch1, sizeof(hipfftDoubleComplex)*NX, NY);
	hipMemcpy2D(dData,pitch1,data,sizeof(hipfftDoubleComplex)*NX,sizeof(hipfftDoubleComplex)*NX,NX,hipMemcpyHostToDevice);
 	
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}

	if (FFT_Z2Z(dData, NX, NY) == -1) { return -1; }
	hipMemcpy(data, dData, sizeof(hipfftDoubleComplex)*NX*NY, hipMemcpyDeviceToHost);
}	
	
	MPI_Bcast(data, NX*NY, MPI_DOUBLE_COMPLEX, 0, MPI_COMM_WORLD);

	// W przypadku Bcast - pozostale watki musza skopiowac data (aka u_in) na GPU
if(ip != 0){
	hipMallocPitch(&dData, &pitch1, sizeof(hipfftDoubleComplex)*NX, NY);
	hipMemcpy2D(dData,pitch1,data,sizeof(hipfftDoubleComplex)*NX,sizeof(hipfftDoubleComplex)*NX,NX,hipMemcpyHostToDevice);

	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}
}


// Liczenie h_z

	hipfftDoubleComplex* h_z_tab;
	h_z_tab = (hipfftDoubleComplex *) malloc ( sizeof(hipfftDoubleComplex)* NX * NY);
	h_z(lam, z, k, sampling, NX, NY, h_z_tab);	

// --- Liczenie H_Z = FFT{h_z_tab} --- //
	
	hipfftDoubleComplex* H_Z;
	hipMalloc((void **) &H_Z, sizeof(hipfftDoubleComplex)* NX * NY);

	size_t pitch2;
 	hipMallocPitch(&H_Z, &pitch2, sizeof(hipfftDoubleComplex)*NX, NY);
	hipMemcpy2D(H_Z,pitch2,h_z_tab,sizeof(hipfftDoubleComplex)*NX,sizeof(hipfftDoubleComplex)*NX,NX,hipMemcpyHostToDevice);
 	
	if (hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda error: Failed to allocate\n");
		return -1;	
	}

	if (FFT_Z2Z(H_Z, NX, NY) == -1) { return -1; }

	// Do the actual multiplication

	multiplyElementwise<<<NX*NY, 1>>>(dData, H_Z, NX*NY);
	

// --- Liczenie u_out = iFFT{dData = U_OUT} --- //

	if (IFFT_Z2Z(dData, NX, NY) == -1) { return -1; }


	hipMemcpy(data, dData, sizeof(hipfftDoubleComplex)*NX*NY, hipMemcpyDeviceToHost);

	//printf( "\nCUFFT vals: \n");
	
// Czytanie calosci


// --- ROLL cwiartek, zeby wszystko sie zgadzalo na koniec --- //

	hipfftDoubleComplex* u_out;
	u_out = (hipfftDoubleComplex *) malloc (sizeof(hipfftDoubleComplex)* NX/2 * NY/2);

	Q_roll(u_out, data, NX, NY);

// --- Przeliczanie Amplitudy --- //

	char filename[128];
	snprintf ( filename, 128, "result_z_%.5lf.txt", z );
	FILE* fp = fopen(filename,"w");

	amplitude_print(u_out, NX, NY, fp);

	fclose(fp);

	endtime = MPI_Wtime();
	printf("\nProces MPI %i: Liczylo sie %f sekund\n", ip, endtime-starttime);
	//MPI_Barrier(MPI_COMM_WORLD);

	hipFree(u_out);
	hipFree(data);
	hipFree(dData);
	hipFree(h_z_tab);
	hipFree(H_Z);
	
	MPI_Finalize();
	
	return 0;
} 
